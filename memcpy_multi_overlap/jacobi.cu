#include "hip/hip_runtime.h"
#include <omp.h>

#include <algorithm>
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <sstream>
#include <string>
#ifdef HAVE_CUB
#include <cub/block/block_reduce.cuh>
#endif  // HAVE_CUB

// From NVIDIA repo
#ifdef USE_NVTX
#include <nvtx3/nvToolsExt.h>

const uint32_t colors[]
    = {0x0000ff00, 0x000000ff, 0x00ffff00, 0x00ff00ff, 0x0000ffff, 0x00ff0000, 0x00ffffff};
const int num_colors = sizeof(colors) / sizeof(uint32_t);

#define PUSH_RANGE(name, cid)                              \
    {                                                      \
        int color = cid;                                   \
        color = color_id % num_colors;                     \
        nvtxEventAttributes_t eventAttrib = {0};           \
        eventAttrib.version = NVTX_VERSION;                \
        eventAttrib.size = NVTX_EVENT_ATTRIB_STRUCT_SIZE;  \
        eventAttrib.colorType = NVTX_COLOR_ARGB;           \
        eventAttrib.color = colors[color_id];              \
        eventAttrib.messageType = NVTX_MESSAGE_TYPE_ASCII; \
        eventAttrib.message.ascii = name;                  \
        nvtxRangePushEx(&eventAttrib);                     \
    }
#define POP_RANGE nvtxRangePop();
#else
#define PUSH_RANGE(name, cid)
#define POP_RANGE
#endif

// Generic argument parsing functions
template <typename T>
T get_argval(char** begin, char** end, const std::string& arg, const T default_val) {
    T val = default_val;
    char** itr = std::find(begin, end, arg);
    if (itr != end && ++itr != end) {
        std::istringstream inbuf(*itr);
        inbuf >> val;
    }
    return val;
}

bool get_argbool(char** begin, char** end, const std::string& arg) {
    char** itr = std::find(begin, end, arg);
    return (itr != end);
}

// CUDA function wrapper to help with debugging. From NVIDIA repo
#define CUDA_RT_CALL(call)                                                                  \
    {                                                                                       \
        hipError_t cudaStatus = call;                                                      \
        if (hipSuccess != cudaStatus) {                                                    \
            fprintf(stderr,                                                                 \
                    "ERROR: CUDA RT call \"%s\" in line %d of file %s failed "              \
                    "with "                                                                 \
                    "%s (%d).\n",                                                           \
                    #call, __LINE__, __FILE__, hipGetErrorString(cudaStatus), cudaStatus); \
            exit(cudaStatus);                                                               \
        }                                                                                   \
    }

constexpr float PI = M_PI;
constexpr float tol = 1.0e-8;
constexpr int MAX_NUM_DEV = 4;  // cluster size @ vast.ai

__global__ void initialize_bounds(float* __restrict__ const a_new, float* __restrict__ const a,
                                  const float pi, const int offset, const int nx, const int my_ny,
                                  const int ny) {
    for (int iy = blockIdx.x * blockDim.x + threadIdx.x; iy < my_ny; iy += blockDim.x * gridDim.x) {
        const float y0 = sin(2.0 * pi * (offset + iy) / (ny - 1));
        a[iy * nx] = y0;
        a[iy * nx + (nx - 1)] = y0;
        a_new[iy * nx] = y0;
        a_new[iy * nx + (nx - 1)] = y0;
    }
}

template <int BLOCK_DIM_X, int BLOCK_DIM_Y>
__global__ void jacobi_kernel(float* __restrict__ const a_new, const float* __restrict__ const a,
                              float* __restrict__ const l2_norm, const int iy_start,
                              const int iy_end, const int nx, const bool calculate_norm) {
#ifdef HAVE_CUB
    typedef hipcub::BlockReduce<float, BLOCK_DIM_X, hipcub::BLOCK_REDUCE_WARP_REDUCTIONS, BLOCK_DIM_Y>
        BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;
#endif  // HAVE_CUB
    int iy = blockIdx.y * blockDim.y + threadIdx.y + iy_start;
    int ix = blockIdx.x * blockDim.x + threadIdx.x + 1;
    float local_l2_norm = 0.0;

    if (iy < iy_end && ix < (nx - 1)) {
        const float val_new = 0.25
                              * (a[iy * nx + ix + 1] + a[iy * nx + ix - 1] + a[(iy + 1) * nx + ix]
                                 + a[(iy - 1) * nx + ix]);
        a_new[nx * iy + ix] = val_new;

        if (calculate_norm) {
            float residue = val_new - a[iy * nx + ix];
            local_l2_norm += residue * residue;
        }
    }
    if (calculate_norm) {
#ifdef HAVE_CUB
        float block_l2_norm = BlockReduce(temp_storage).Sum(local_l2_norm);
        if (0 == threadIdx.y && 0 == threadIdx.x) atomicAdd(l2_norm, block_l2_norm);
#else
        atomicAdd(l2_norm, local_l2_norm);
#endif  // HAVE_CUB
    }
}

double single_gpu(const int nx, const int ny, const int iter_max, float* const a_ref_h,
                  const int nccheck, const bool debug) {
    // DONE: Finish this impl
    float* a;
    float* a_new;

    hipStream_t compute_stream;
    hipStream_t push_top_stream;
    hipStream_t push_bottom_stream;
    hipEvent_t compute_done;
    hipEvent_t push_top_done;
    hipEvent_t push_bottom_done;

    float* l2_norm_d;
    float* l2_norm_h;

    int iy_start = 1;
    int iy_end = (ny - 1);

    CUDA_RT_CALL(hipMalloc(&a, nx * ny * sizeof(float)));
    CUDA_RT_CALL(hipMalloc(&a_new, nx * ny * sizeof(float)));
    CUDA_RT_CALL(hipMemset(a, 0, nx * ny * sizeof(float)));
    CUDA_RT_CALL(hipMemset(a_new, 0, nx * ny * sizeof(float)));

    // diriclet boundaries
    initialize_boundaries<<<ny / 128 + 1, 128>>>(a, a_new, PI, 0, nx, ny, ny);
    CUDA_RT_CALL(hipGetLastError());
    CUDA_RT_CALL(hipDeviceSynchronize());
    CUDA_RT_CALL(hipStreamCreate(&compute_stream));
    CUDA_RT_CALL(hipStreamCreate(&push_top_stream));
    CUDA_RT_CALL(hipStreamCreate(&push_bottom_stream));
    CUDA_RT_CALL(hipEventCreateWithFlags(&compute_done, hipEventDisableTiming));
    CUDA_RT_CALL(hipEventCreateWithFlags(&push_top_done, hipEventDisableTiming));
    CUDA_RT_CALL(hipEventCreateWithFlags(&push_bottom_done, hipEventDisableTiming));
    CUDA_RT_CALL(hipMalloc(&l2_norm_d, sizeof(float)));
    CUDA_RT_CALL(hipHostMalloc(&l2_norm_h, sizeof(float)));
    CUDA_RT_CALL(hipDeviceSynchronize());

    constexpr int dim_block_x = 32;
    constexpr int dim_block_y = 32;
    dim3 dim_grid((nx + dim_block_x - 1) / dim_block_x, (ny + dim_block_y - 1) / dim_block_y, 1);

    bool calculate_norm = true;
    int iter = 0;
    float l2_norm = 1.0;
    double start = omp_get_wtime();
    PUSH_RANGE("Jacobi solve", 0)
    while (l2_norm > tol && iter < iter_max) {
        CUDA_RT_CALL(hipMemsetAsync(l2_norm_d, 0, sizeof(float), compute_stream));
        CUDA_RT_CALL(hipStreamWaitEvent(compute_stream, push_top_done, 0));
        CUDA_RT_CALL(hipStreamWaitEvent(compute_stream, push_bottom_done, 0));
        calculate_norm = (iter % nccheck) == 0 || (print && ((iter % 100) == 0));
        jacobi_kernel<dim_block_x, dim_block_y>
            <<<dim_grid, {dim_block_x, dim_block_y, 1}, 0, compute_stream>>>(
                a_new, a, l2_norm_d, iy_start, iy_end, nx, calculate_norm);
        CUDA_RT_CALL(hipGetLastError());
        CUDA_RT_CALL(hipEventRecord(compute_done, compute_stream));

        if (calculate_norm)
            CUDA_RT_CALL(hipMemcpyAsync(l2_norm_h, l2_norm_d, sizeof(float),
                                         hipMemcpyDeviceToHost, compute_stream));

        CUDA_RT_CALL(hipStreamWaitEvent(push_top_stream, compute_done, 0));
        CUDA_RT_CALL(hipMemcpyAsync(a_new, a_new + (iy_end - 1) * nx, nx * sizeof(float),
                                     hipMemcpyDeviceToDevice, push_top_stream));
        CUDA_RT_CALL(hipEventRecord(push_top_done, push_top_stream));
        CUDA_RT_CALL(hipStreamWaitEvent(push_bottom_stream, compute_done, 0));
        CUDA_RT_CALL(hipMemcpyAsync(a_new + iy_end * nx, a_new + iy_start * nx, nx * sizeof(float),
                                     hipMemcpyDeviceToDevice, compute_stream));
        CUDA_RT_CALL(hipEventRecord(push_bottom_done, push_bottom_stream));

        if (calculate_norm) {
            CUDA_RT_CALL(hipStreamSynchronize(compute_stream));
            l2_norm = *l2_norm_h;
            l2_norm = std::sqrt(l2_norm);
            if (debug && (iter % 100) == 0) printf("%5d, %0.6f\n", iter, l2_norm);
        }

        std::swap(a_new, a);
        iter++;
    }
    POP_RANGE
    double stop = omp_get_wtime();

    CUDA_RT_CALL(hipMemcpy(a_ref_h, a, nx * ny * sizeof(float), hipMemcpyDeviceToHost));
    CUDA_RT_CALL(hipEventDestroy(push_bottom_done));
    CUDA_RT_CALL(hipEventDestroy(push_top_done));
    CUDA_RT_CALL(hipEventDestroy(compute_done));
    CUDA_RT_CALL(hipStreamDestroy(push_bottom_stream));
    CUDA_RT_CALL(hipStreamDestroy(push_top_stream));
    CUDA_RT_CALL(hipStreamDestroy(compute_stream));
    CUDA_RT_CALL(hipHostFree(l2_norm_h));
    CUDA_RT_CALL(hipFree(l2_norm_d));
    CUDA_RT_CALL(hipFree(a_new));
    CUDA_RT_CALL(hipFree(a));
    return (stop - start);
    // return 0.0
}

int main(int argc, char* argv[]) {
    // Parse command line arguments. add anything else?
    const int iter_max = get_argval<int>(argv, argv + argc, "-niter", 1000);
    const bool csv = get_argbool(argv, argv + argc, "-csv");
    const int nx = get_argval<int>(argv, argv + argc, "-nx", 100);
    const int ny = get_argval<int>(argv, argv + argc, "-ny", 100);
    const bool nop2p = get_argbool(argv, argv + argc, "-nop2p");
    const int nccheck = get_argval<int>(argv, argv + argc, "-nccheck", 1);

    // float *a[MAX_NUM_DEV],
    float* a_new[MAX_NUM_DEV];
    float *a_h, *a_ref_h;
    double serial_runtime = 0.0;

    hipEvent_t push_top_done[2][MAX_NUM_DEV];
    hipEvent_t push_bottom_done[2][MAX_NUM_DEV];

    // int iy_start[MAX_NUM_DEV];
    int iy_end[MAX_NUM_DEV];
    // int chunk_size[MAX_NUM_DEV];
    // float* l2_norm_d[MAX_NUM_DEV];
    // float* l2_norm_h[MAX_NUM_DEV];
    float l2_norm = 1.0;
    bool result_correct = true;
    int num_dev = 0;
    CUDA_RT_CALL(hipGetDeviceCount(&num_dev));
#pragma opm parallel num_threads(num_dev) shared(l2_norm)
    {
        float* a;
        hipStream_t compute_stream;
        hipStream_t push_top_stream;
        hipStream_t push_bottom_stream;
        hipEvent_t compute_done;
        float* l2_norm_d;
        float* l2_norm_h;
        int dev_id = omp_get_thread_num();

        CUDA_RT_CALL(hipSetDevice(dev_id));
        CUDA_RT_CALL(hipFree(0));  // reset device memory
        if (dev_id == 0) {
            CUDA_RT_CALL(hipHostMalloc(&a_h, nx * ny * sizeof(float)));
            CUDA_RT_CALL(hipHostMalloc(&a_ref_h, nx * ny * sizeof(float)));
            serial_runtime = single_gpu(nx, ny, iter_max, a_ref_h, nccheck, !csv);
        }
#pragma omp barrier

        // optimize load balancing through row dist.
        int chunk_size;
        int chunk_size_low = (ny - 2) / num_devices;
        int num_ranks_low = num_devices * chunk_size_low + num_devices - (ny - 2);
        int chunk_size_high = chunk_size_low + 1;
        chunk_size[dev_id] = (dev_id < num_ranks_low) ? chunk_size_low : chunk_size_high;
        CUDA_RT_CALL(hipMalloc(&a, nx * (chunk_size + 2) * sizeof(float)));
        CUDA_RT_CALL(hipMalloc(a_new + dev_id, nx * (chunk_size + 2) * sizeof(float)));
        CUDA_RT_CALL(hipMemset(a, 0, nx * (chunk_size + 2) * sizeof(float)));
        CUDA_RT_CALL(hipMemset(a_new[dev_id], 0, nx * (chunk_size + 2) * sizeof(float)));

        // Calculate local domain boundaries
        int iy_start_global;
        if (dev_id < num_ranks_low) {
            iy_start_global = dev_id * chunk_size_low + 1;
        } else {
            iy_start_global
                = num_ranks_low * chunk_size_low + (dev_id - num_ranks_low) * chunk_size_high + 1;
        }

        iy_start = 1;
        iy_end[dev_id] = iy_start + chunk_size;

        // TODO: set boundaries on left/right
        initialize_bounds<<<(ny / num_devices) / 128 + 1, 128>>>(
            a, a_new[dev_id], PI, iy_start_global - 1, nx, (chunk_size + 2), ny);

        CUDA_RT_CALL(hipGetLastError());
        CUDA_RT_CALL(hipDeviceSynchronize());

        int leastPriority = 0;
        int greatestPriority = leastPriority;
        CUDA_RT_CALL(hipDeviceGetStreamPriorityRange(&leastPriority, &greatestPriority));

        CUDA_RT_CALL(
            hipStreamCreateWithPriority(&compute_stream, hipStreamDefault, leastPriority));
        CUDA_RT_CALL(
            hipStreamCreateWithPriority(&push_top_stream, hipStreamDefault, greatestPriority));
        CUDA_RT_CALL(
            hipStreamCreateWithPriority(&push_bottom_stream, hipStreamDefault, greatestPriority));
        //
        // CUDA_RT_CALL(hipStreamCreate(&compute_stream));
        // CUDA_RT_CALL(hipStreamCreate(&push_top_stream));
        // CUDA_RT_CALL(hipStreamCreate(&push_bottom_stream));
        //
        // CUDA_RT_CALL(hipEventCreateWithFlags(&compute_done, hipEventDisableTiming));
        CUDA_RT_CALL(hipEventCreateWithFlags(push_top_done[0] + dev_id, hipEventDisableTiming));
        CUDA_RT_CALL(
            hipEventCreateWithFlags(push_bottom_done[0] + dev_id, hipEventDisableTiming));
        CUDA_RT_CALL(hipEventCreateWithFlags(push_top_done[1] + dev_id, hipEventDisableTiming));
        CUDA_RT_CALL(
            hipEventCreateWithFlags(push_bottom_done[1] + dev_id, hipEventDisableTiming));

        CUDA_RT_CALL(hipMalloc(&l2_norm_d, sizeof(float)));
        CUDA_RT_CALL(hipHostMalloc(&l2_norm_h, sizeof(float)));

        if (!nop2p) {
            const int top = dev_id > 0 ? dev_id - 1 : (num_devices - 1);
            int canAccessPeer = 0;

            CUDA_RT_CALL(hipDeviceCanAccessPeer(&canAccessPeer, dev_id, top));

            if (canAccessPeer) CUDA_RT_CALL(hipDeviceEnablePeerAccess(top, 0));
            const int bottom = (dev_id + 1) % num_devices;
            if (top != bottom) {
                canAccessPeer = 0;
                CUDA_RT_CALL(hipDeviceCanAccessPeer(&canAccessPeer, dev_id, bottom));
                if (canAccessPeer) CUDA_RT_CALL(hipDeviceEnablePeerAccess(bottom, 0));
            }
        }
        // CUDA_RT_CALL(hipDeviceSynchronize());
        // }

        for (int i = 0; i < 4; ++i) {
            // for (int dev_id = 0; dev_id < num_devices; ++dev_id) {
            // CUDA_RT_CALL(hipSetDevice(dev_id));
            // const int top = dev_id > 0 ? dev_id - 1 : (num_devices - 1);
            // const int bottom = (dev_id + 1) % num_devices;
            CUDA_RT_CALL(hipMemcpyAsync(a_new[top] + (iy_end[top] * nx),
                                         a_new[dev_id] + iy_start * nx, nx * sizeof(real),
                                         hipMemcpyDeviceToDevice, push_top_stream));
            CUDA_RT_CALL(hipMemcpyAsync(a_new[bottom], a_new[dev_id] + (iy_end[dev_id] - 1) * nx,
                                         nx * sizeof(real), hipMemcpyDeviceToDevice,
                                         push_bottom_stream));
            // }
            // for (int dev_id = 0; dev_id < num_devices; ++dev_id) {
            // CUDA_RT_CALL(hipSetDevice(dev_id));
            CUDA_RT_CALL(hipStreamSynchronize(push_top_stream));
            CUDA_RT_CALL(hipStreamSynchronize(push_bottom_stream));
            // }
            // for (int dev_id = 0; dev_id < num_devices; ++dev_id) {
            std::swap(a_new[dev_id], a[dev_id]);
            // }
        }
        CUDA_RT_CALL(hipDeviceSynchronize());

#pragma opm master
        {
            if (!csv) printf("norm\n");
        }

        constexpr int dim_block_x = 32;
        constexpr int dim_block_y = 32;
        dim3 dim_grid((nx + dim_block_x - 1) / dim_block_x,
                      (ny + (num_devices * dim_block_y) - 1) / (num_devices * dim_block_y), 1);
        int iter = 0;
        bool calculate_norm = true;
#pragma omp master
        {
            real l2_norm = 1.0;
        }
        CUDA_RT_CALL(hipDeviceSynchronize());

#pragma omp barrier

        // for (int dev_id = 0; dev_id < num_devices; ++dev_id) {
        //     CUDA_RT_CALL(hipSetDevice(dev_id));
        //     CUDA_RT_CALL(hipDeviceSynchronize());
        // }
        double start = omp_get_wtime();
        PUSH_RANGE("Jacobi solve", 0)
        while (l2_norm > tol && iter < iter_max) {
            // for (int dev_id = 0; dev_id < num_devices; ++dev_id) {
            CUDA_RT_CALL(hipMemsetAsync(l2_norm_d, 0, sizeof(real), compute_stream));

            const int top = dev_id > 0 ? dev_id - 1 : (num_devices - 1);
            const int bottom = (dev_id + 1) % num_devices;
            // CUDA_RT_CALL(hipSetDevice(dev_id));

            CUDA_RT_CALL(hipStreamWaitEvent(compute_stream, push_top_done[(iter % 2)][bottom], 0));
            CUDA_RT_CALL(hipStreamWaitEvent(compute_stream, push_bottom_done[(iter % 2)][top], 0));

            calculate_norm = (iter % nccheck) == 0 || (!csv && (iter % 100) == 0);
            // dim3 dim_grid((nx + dim_block_x - 1) / dim_block_x,
            // (chunk_size[dev_id] + dim_block_y - 1) / dim_block_y, 1);

            jacobi_kernel<dim_block_x, dim_block_y>
                <<<dim_grid, {dim_block_x, dim_block_y, 1}, 0, compute_stream[dev_id]>>>(
                    a_new[dev_id], a, l2_norm_d, iy_start, iy_end[dev_id], nx, calculate_norm);
            CUDA_RT_CALL(hipGetLastError());
            CUDA_RT_CALL(hipEventRecord(compute_done, compute_stream));

            if (calculate_norm)
                CUDA_RT_CALL(hipMemcpyAsync(l2_norm_h[dev_id], l2_norm_d[dev_id], sizeof(real),
#pragma omp barrier
            // periodic boundary conds
            CUDA_RT_CALL(hipStreamWaitEvent(push_top_stream, compute_done, 0));
            CUDA_RT_CALL(hipMemcpyAsync(a_new[top] + (iy_end[top] * nx),
                                         a_new + iy_start * nx, nx * sizeof(real),
                                         hipMemcpyDeviceToDevice, push_top_stream));
            CUDA_RT_CALL(
                hipEventRecord(push_top_done[((iter + 1) % 2)][dev_id], push_top_stream));

            CUDA_RT_CALL(hipStreamWaitEvent(push_bottom_stream, compute_done, 0));
            CUDA_RT_CALL(hipMemcpyAsync(a_new[bottom], a_new[dev_id] + (iy_end[dev_id] - 1) * nx,
                                         nx * sizeof(real), hipMemcpyDeviceToDevice,
                                         push_bottom_stream));
            CUDA_RT_CALL(hipEventRecord(push_bottom_done[((iter + 1) % 2)][dev_id],
                                         push_bottom_stream));
        }
#pragma omp barrier
        if (calculate_norm) {
#pragma omp single
            {
                l2_norm = 0.0;
            }
#pragma omp barrier
            // for (int dev_id = 0; dev_id < num_devices; ++dev_id) {
            CUDA_RT_CALL(hipStreamSynchronize(compute_stream));
#pragma omp atomic
            l2_norm += *(l2_norm_h);
            // }
#pragma omp barrier
#pragma opm single
            {
                l2_norm = std::sqrt(l2_norm);
            }
#pragma omp barrier
            if (!csv && (iter % 100) == 0) printf("%5d, %0.6f\n", iter, l2_norm);
        }

#pragma omp master
        // for (int dev_id = 0; dev_id < num_devices; ++dev_id) {
        std::swap(a_new[dev_id], a);
        // }
        iter++;
    }
    // for (int dev_id = 0; dev_id < num_devices; ++dev_id) {
    // CUDA_RT_CALL(hipSetDevice(dev_id));
    CUDA_RT_CALL(hipDeviceSynchronize());
    // }
#pragma omp barrier
    double stop = omp_get_wtime();
    POP_RANGE

    // int offset = nx;
    // for (int dev_id = 0; dev_id < num_devices; ++dev_id) {
    CUDA_RT_CALL(hipMemcpy(a_h + iy_start_global * nx, a + nx,
                            std::min((ny - iy_start_global) * nx, chunk_size * nx) * sizeof(float),
                            hipMemcpyDeviceToHost));
    // CUDA_RT_CALL(
    // hipMemcpy(a_h + offset, a + nx,
    // std::min((nx * ny) - offset, nx * chunk_size[dev_id]) * sizeof(real),
    // hipMemcpyDeviceToHost));
    // offset += std::min(chunk_size[dev_id] * nx, (nx * ny) - offset);
#pragma omp barrier

#pragma omp master
    {
        bool result_correct = true;
        for (int iy = 1; result_correct && (iy < (ny - 1)); ++iy) {
            for (int ix = 1; result_correct && (ix < (nx - 1)); ++ix) {
                if (std::fabs(a_ref_h[iy * nx + ix] - a_h[iy * nx + ix]) > tol) {
                    fprintf(stderr,
                            "ERROR: a[%d * %d + %d] = %f does not match %f "
                            "(reference)\n",
                            iy, nx, ix, a_h[iy * nx + ix], a_ref_h[iy * nx + ix]);
                    result_correct = false;
                }
            }
        }
        if (result_correct) {
            if (csv) {
                printf("single_threaded_copy, %d, %d, %d, %d, %d, %d, %f, %f\n", nx, ny, iter_max,
                       nccheck, num_devices, nop2p ? 0 : 1, (stop - start), runtime_serial);
            } else {
                printf("Num GPUs: %d.\n", num_devices);
                printf(
                    "%dx%d: 1 GPU: %8.4f s, %d GPUs: %8.4f s, speedup: %8.2f, "
                    "efficiency: %8.2f \n",
                    ny, nx, runtime_serial, num_devices, (stop - start),
                    runtime_serial / (stop - start),
                    runtime_serial / (num_devices * (stop - start)) * 100);
            }
        }
    }  // omp master

    // for (int dev_id = (num_devices - 1); dev_id >= 0; --dev_id) {
    // CUDA_RT_CALL(hipSetDevice(dev_id));
    CUDA_RT_CALL(hipEventDestroy(push_bottom_done[1][dev_id]));
    CUDA_RT_CALL(hipEventDestroy(push_top_done[1][dev_id]));
    CUDA_RT_CALL(hipEventDestroy(push_bottom_done[0][dev_id]));
    CUDA_RT_CALL(hipEventDestroy(push_top_done[0][dev_id]));
    CUDA_RT_CALL(hipEventDestroy(compute_done));
    CUDA_RT_CALL(hipStreamDestroy(push_bottom_stream));
    CUDA_RT_CALL(hipStreamDestroy(push_top_stream));
    CUDA_RT_CALL(hipStreamDestroy(compute_stream));

    CUDA_RT_CALL(hipHostFree(l2_norm_h));
    CUDA_RT_CALL(hipFree(l2_norm_d));

    CUDA_RT_CALL(hipFree(a_new[dev_id]));
    CUDA_RT_CALL(hipFree(a));
    if (dev_id == 0) {
        CUDA_RT_CALL(hipHostFree(a_h));
        CUDA_RT_CALL(hipHostFree(a_ref_h));
    }
}

return !result_correct;
}
